#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include <corecrt_math_defines.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <tuple>
#include <vector>
#include "pgm.h"
#include "CImg.h"

using namespace std;

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, vector<tuple<int, int, int, int, int>> *acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  //*acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  //memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  // inicializar el acumulador en ceros
  //*acc = vector<tuple<int, int, int>>(rBins * degreeBins, make_tuple(0, 0, 0));
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  //cout << "rMax: " << rMax << endl;
  //cout << "xCent: " << xCent << endl;
  //cout << "yCent: " << yCent << endl;
  //cout << "rScale: " << rScale << endl;

  bool continueloop = true;

  for (int i = 0; i < w; i++) { //por cada pixel
    for (int j = 0; j < h; j++) //...
    {
      int idx = j * w + i;
      if (pic[idx] > 10 && continueloop) //si pasa thresh, entonces lo marca
      {
        int xCoord = i - xCent;
        int yCoord = yCent - j;  // y-coord has to be reversed
        //float theta = 0;         // actual angle
        //cout << "xCoord: " << xCoord << endl;
        //cout << "yCoord: " << yCoord << endl;
        // recorre de 0 a pi, pero por indices
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
        {
          //cout << "------------------------------------" << endl;
          float theta = tIdx * radInc;
            
          //cout << "theta: " << theta << endl;
          float r = xCoord * cos (theta) + yCoord * sin (theta);
          //cout << "r: " << r << endl;
          //xint rIdx = (r + rMax) / rScale;
          int rIdx = static_cast<int>((r + rMax) / rScale);
          //float rIdx = (r + rMax) / rScale;
          //cout << "rIdx: " << rIdx << endl;
          //cout << "acc index: " << rIdx * degreeBins + tIdx << endl;
          // +1 para este radio r y este theta
          acc->at(rIdx * degreeBins + tIdx) = make_tuple(rIdx, tIdx, xCoord, yCoord, get<4>(acc->at(rIdx * degreeBins + tIdx)) + 1);
          //cout << "acc: " << get<4>(acc->at(rIdx * degreeBins + tIdx)) << endl;
          //xtheta += radInc;
          //x//cout << "theta: " << theta << endl;
        }
        /*if (i == (w/2)) {
          continueloop = false;
        }*/
        //continueloop = false;
      }
    }
  }
}
bool compareTuples(const tuple<int, int, int, int, int>& a, const tuple<int, int, int, int, int>& b) {
    return get<4>(a) > get<4>(b); // Sort in descending order
}

//*****************************************************************
// TODO usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
//__constant__ float d_Cos[degreeBins];
//__constant__ float d_Sin[degreeBins];

//*****************************************************************
//TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }
//TODO Kernel memoria Constante
// __global__ void GPU_HoughTranConst(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados

}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  vector<tuple<int, int, int, int, int>> *cpuht = new vector<tuple<int, int, int, int, int>>(degreeBins * rBins, make_tuple(0, 0, 0, 0, 0));
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  //cout << "size: " << w << " x " << h << endl;

  float* d_Cos;
  float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  //cout << "cpuht size: " << cpuht->size() << endl;
  //cout << "quick view of cpuht: " << endl;
  for (i = 0; i < 10; i++)
  {
    //cout << get<0>(cpuht->at(i)) << " " << get<1>(cpuht->at(i)) << " " << get<2>(cpuht->at(i)) << endl;
  }

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, cpuht);

  //////////////////////////////////////////////////// test
  vector<tuple<int, int, int, int, int>> lines; // posX0, posY0, posX1, posY1
  vector<tuple<int, int, int, int>> linesTop; // posX0, posY0, posX1, posY1

  int threshold = 50;
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  //cout << "Empieza a buscar lineas" << endl;
  for (i = 0; i < degreeBins * rBins; i++)
  {
    // cout << "vote at: " << i << ": " << get<4>(cpuht->at(i)) << endl;
    if (get<4>(cpuht->at(i)) > threshold)
    {
      //cout << "------------------------------------" << endl;
      //cout << "indice: " << i << endl;
      // convierte  theta de radianes a grados
      float theta = get<1>(cpuht->at(i)) * degreeInc * M_PI / 180;
      //cout << "theta: " << theta << endl;
      // convierte r de indice a valor real
      float r = (get<0>(cpuht->at(i)) * 2 * rMax / rBins) - rMax;
      //cout << "r: " << r << endl;
      // calcula los puntos de la linea
      // si theta es 0, entonces es una linea vertical
      if (theta == 0.0)
      {
        int personalVote = static_cast<int>(get<4>(cpuht->at(i)));
        lines.push_back(make_tuple(static_cast<int>(r)+(w/2), 0, static_cast<int>(r+(w/2)), h, personalVote));
      }
      else
      {
        // sin tomar en cuenta limites de la imagen
        float m = round(-cos(theta) / sin(theta));
        float b = r / sin(theta);

        if (m == 0.0) { // horizontal line
          float b = r / sin(theta);

          int x0 = get<2>(cpuht->at(i)) + (w/2);
          int y0 = b;
          int x1 = get<2>(cpuht->at(i)) + (w/2);
          int y1 = static_cast<int>(b);

          int personalVote = static_cast<int>(get<4>(cpuht->at(i)));
          lines.push_back(make_tuple(x0, y0, x1, y1, personalVote));

          x0 = get<2>(cpuht->at(i)) + (w/2);
          y0 = static_cast<int>(x0 + b);
          y1 = static_cast<int>(b);
          x1 = get<2>(cpuht->at(i)) + (w/2);
          

          lines.push_back(make_tuple(x0, y0, x1, y1, personalVote));


          continue;
        }
        //cout << "m: " << m << endl;
        //cout << "b: " << b << endl;
        int x0 = get<2>(cpuht->at(i)) + (w/2);
        int y0 = -get<3>(cpuht->at(i)) + (h/2);
        int x1 = w;
        int y1 = static_cast<int>((m * x1 + b));
        // con limites de la imagen
        int personalVote = static_cast<int>(get<4>(cpuht->at(i)));
        lines.push_back(make_tuple(x0, y0, x1, y1, personalVote));
        // lineas del otro lado
        // cout << " line 1 " << x0 << " " << y0 << " " << x1 << " " << y1 << endl;
        x0 = get<2>(cpuht->at(i)) + (w/2);
        y0 = -get<3>(cpuht->at(i)) + (h/2);
 
        y1 = static_cast<int>((m * x0 + b));
        cout << "pendiente negativa"<< m;
        x1 = static_cast<int>(((y1 - b)/m));
        // cout << " line 2 " << x0 << " " << y0 << " " << x1 << " " << y1 << endl;

        lines.push_back(make_tuple(x0, y0, x1, y1, personalVote));
      }
    }
  }
  /////////////////////////////////////////////////////////
  //cimg_library::CImg<unsigned char> image("./cuadrosHough.pgm");
  cimg_library::CImg<unsigned char> image(w, h, 1, 3, 255);
  
  if (image.is_empty()) {
      std::cout << "Could not open or find the image." << std::endl;
      return -1;
  }
  
  // Draw lines on the image
  //int x0 = 100, y0 = 100, x1 = 200, y1 = 200;
  const unsigned char red[] = { 255,0,0 };
  const unsigned char black[] = { 0,0,0 };

  const float opacity = 1;
  //const unsigned int pattern = ~0U;
  //image.draw_line(x0,y0,x1,y1,red,opacity);
 

  sort(lines.begin(), lines.end(), compareTuples);
  for (const auto& tuple : lines) {
      cout << "Valor: " << get<4>(tuple) << endl;
  }

  for (size_t i = 0; i < 4; i++)
  {
      cout << " line 1 " << get<0>(lines[i]) << " " << get<1>(lines[i]) << " " << get<2>(lines[i]) << " " << get<3>(lines[i]) << endl;
    image.draw_line(get<0>(lines[i]), get<1>(lines[i]), get<2>(lines[i]), get<3>(lines[i]), red, opacity);
  }
  
  // image.draw_line(42, 141, 0, -1701, black, opacity);

  // for (auto line : lines)
  // {
  //   image.draw_line(get<0>(line), get<1>(line), get<2>(line), get<3>(line), red, opacity);
  // }

  /*int j;
  for (i = 0; i < w; i++) {
    for (j = 0; j < h; j++) {
      int idx = j * w + i;
      ////cout << (int)inImg.pixels[idx] << " ";
      if (inImg.pixels[idx] >= 50) {
        image(i, j, 0) = 0;
        image(i, j, 1) = 255;
        image(i, j, 2) = 0;
      }
    }
  }*/

  // Save the modified image to a new file
  image.save("test.bmp");
  
  std::cout << "Image processing complete." << std::endl;
  //////////////////////////////////////////////////////

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  //float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global
  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);
  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  /*for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }*/
  printf("Done!\n");

  // imprimir pixeles en imagen
  PGMImage outImg (degreeBins, rBins, 1);
  outImg.pixels = (unsigned char *) h_hough;
  outImg.write(argv[2]);

  // TODO clean-up
  
  return 0;
}
